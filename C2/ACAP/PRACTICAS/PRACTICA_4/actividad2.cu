
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>

#define NBLOCKS 8192
#define THREADS_PER_BLOCK 256

const int numRuns = 10;

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

void vector_init(double* vector, int size){
    for(int i=0; i < size; i++){
        vector[i] = (double)rand()/RAND_MAX + rand();
    }
}

__device__ double MyatomicMax(double *address, double val)
{
    unsigned long long ret = __double_as_longlong(*address);
    while(val > __longlong_as_double(ret))
    {
        unsigned long long old = ret;
        if((ret = atomicCAS((unsigned long long *)address, old, __double_as_longlong(val))) == old)
            break;
    }
    return __longlong_as_double(ret);
}

__global__ __forceinline__ void kernelMax(double* vector, int size, double* max){
    __shared__ int workSize;
    __shared__ int offset;
    __shared__ int start;
    __shared__ int quantity;
    __shared__ double max_local;

    if(threadIdx.x == 0){
        workSize = size/gridDim.x;
        offset = size%gridDim.x;
        start = blockIdx.x*workSize + (blockIdx.x < offset?blockIdx.x:offset);
        quantity = workSize + (blockIdx.x < offset);
        max_local = vector[0];
    }

    __syncthreads();

    for(int i=start+threadIdx.x; i < start+quantity; i+=blockDim.x){
        if(max_local < vector[i]){
            MyatomicMax(&max_local, vector[i]);
        }
    }

    __syncthreads();

    if(*max < max_local && threadIdx.x == 0)
       MyatomicMax(max,max_local);
}

double maximo_secuencial(double* vector, int size){
    double inicio, fin;

    inicio = get_wall_time();

    double max = vector[0];

    for(int i=1; i < size; i++){
        if(vector[i] > max)
            max = vector[i];
    }

    fin = get_wall_time();

    printf("El tiempo consumido por la CPU es %f\n", fin-inicio);

    return max;
}

void comparar_maximos(double max_cpu, double max_gpu){
    if(max_cpu != max_gpu)
        printf("Error!!!\n");
    
    else{
        printf("\n---Comprobación correcta!!!---\n");
    }

    printf("Maximo-CPU: %f\n", max_cpu);
    printf("Maximo-GPU: %f\n", max_gpu);
}
int main(int argc, char * argv[]){
    if(argc < 2){
        printf("Usage: ./actividad1.cu <tam_vec>\n");
        exit(EXIT_FAILURE);
    }

    int size = atoi(argv[1]);
    double* vec = (double*)malloc(size*sizeof(double));

    vector_init(vec, size);

    double max_cpu = maximo_secuencial(vec, size);
    double max_gpu = 0.0;

    double *dev_vec = nullptr;
    double *dev_max = nullptr;

    float milliSeconds = 0.0;
    float aveGPUMS = 0.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for(int i=-1; i < numRuns; i++){
        hipEventRecord(start);

        hipMalloc( (void**) &dev_vec, size*sizeof( double ) );
        hipMemcpy( dev_vec, vec, size*sizeof(double), hipMemcpyHostToDevice );
        hipMalloc( (void**) &dev_max, sizeof( double ));
    
        hipMemset(dev_max, 0.0 , sizeof(double));
        
        if(i < 0)
            kernelMax<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_vec, size, dev_max);
        
        else{
            hipDeviceSynchronize();
            kernelMax<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_vec, size, dev_max);
            hipDeviceSynchronize();
            hipMemcpy( &max_gpu, dev_max, sizeof(double), hipMemcpyDeviceToHost);
        }

        hipFree(dev_vec);
        hipFree(dev_max);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliSeconds, start,stop);

        if(i>=0)
            aveGPUMS += milliSeconds;
    }

    printf("El tiempo consumido por la GPU es %f\n", aveGPUMS/(1000*numRuns));

    comparar_maximos(max_cpu, max_gpu);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}