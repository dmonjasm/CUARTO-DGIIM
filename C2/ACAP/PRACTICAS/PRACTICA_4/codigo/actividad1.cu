#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

void vector_init(double* vector, int size){
    for(int i=0; i < size; i++){
        vector[i] = (double)rand()/RAND_MAX;
    }
}

__global__ void calcularMax(double* vector, int size){
    __shared__ double aux;
}

double maximo_secuencial(double* vector, int size){
    double max = vector[0];

    for(int i=1; i < size; i++){
        if(vector[i] > max)
            max = vector[i];
    }

    return max;
}

double comparar_maximos(double max_cpu, double max_gpu){
    if(max_cpu != max_gpu)
        printf("Error!!!\n");
    
    else
        printf("Comprobación correcta\n");
}
int main(int argc, char * argv[]){
    if(argc < 2){
        printf("Usage: ./actividad1.cu <tam_vec>\n");
        exit(EXIT_FAILURE);
    }

    int size = atoi(argv[1]);
    double* vec = (double*)malloc(size*sizeof(double));

    vector_init(vec, size);

    double max_cpu = maximo_secuencial(vec, size);

    double* dev_vec = nullptr;
    hipMalloc3D()

    for(int i=0; i < size; i++)
        printf("%f\n", vec[i]);

    printf("El maximo es: %f\n", max_cpu);
    
    return 0;
}