
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <cmath>

#define NBLOCKS 8192
#define THREADS_PER_BLOCK 256
#define EPSILON 0.000000000001

const int numRuns = 10;

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

void sillyInit(double** matrixA, int filas, int columnas){
    for(int i=0; i < filas; i++){
        for(int j=0; j< columnas; j++){
            matrixA[i][j] = (double) rand()/RAND_MAX;
        }
    }
}

double** reservarMatriz(int fil, int col){
    double** matriz = 0;
    double* matriz_container =(double *) malloc(sizeof(double)*fil*col);//Gran vector contiguo

    if(matriz_container){
        matriz = (double**) malloc(sizeof(double*)*fil);//Aqui guardaremos el comienzo de cada fila

        if(matriz){
            for(int i = 0; i<fil; i++){//Vamos a ir apuntando a cada fila
                matriz[i] = &(matriz_container[i*col]);//El inicio de la fila i es i*columnas
            }
        }
        
        else{//Ahora pediir memoria ha fallado... pero:
            printf("Error. No se ha podido reservar la carcasa 2D de la matriz.\n");
            free(matriz_container);//Esto si funciono, y debemos limpiarlo
        }
    }
    
    else{
        printf("Error. No se ha podido reservar memoria para la matriz.\n");
    }

    return matriz;//Tenemos esta memoria heap... y luego otro debera liberarla eh
}

void liberarMatriz(double** matriz){
	if(matriz){//Liberando el principio de la matriz ya estamos apuntando al principio del container
		free(matriz[0]);//Pero claro, solo si apunta a algo
	}

	free(matriz);//Ahora podemos eliminar el conjunto de punteros a fila
}

void printMatrix(double** matrixA, int filas, int columnas){
    for(int i=0; i < filas; i++){
        for(int j= 0; j < columnas; j++){
            printf("%.3lf ", matrixA[i][j]);
        }

        printf("\n");
    }

    printf("\n\n");
}

double** productoMatrices(double** matrixA, int filA, int colA, double** matrixB, int colB){
    double** matrixC = reservarMatriz(filA, colB);
    double inicio = 0.0, final = 0.0;


    inicio = get_wall_time();
    for(int i=0; i < filA;i++){
        for(int j=0; j < colB;j++){
            matrixC[i][j]=0.0;
            for(int k=0; k < colA ;k++){
                matrixC[i][j] += matrixA[i][k] * matrixB[k][j];
            }
        }
    }

    final = get_wall_time();

    printf("Tiempo usado por la CPU:%f\n", final-inicio);

    return matrixC;
}

void comprobarMatrices(double** matCPU, int fil, int col, double** matGPU){
    bool iguales = true;

    for(int i=0; i < fil && iguales; i++){
        for(int j=0; j < col && iguales; j++){
            if(abs(matCPU[i][j] - matGPU[i][j]) > EPSILON){   
                iguales = false;
                printf("i:%i\tj:%i\n", i, j);
                printf("MatrizCPU:%.9f\n", matCPU[i][j]);
                printf("MatrizGPU:%.9f\n", matGPU[i][j]);
            }
        }
    }

    if(iguales)
        printf("--Cálculo correcto!!!--\n");

    else
        printf("--ERROR--\n");
}

__global__ void kernelProductoMatrices(double* matA, int filA, int colA, double* matB, int colB, double* matC){
    __shared__ int workSize;
    __shared__ int offset;
    __shared__ int start;
    __shared__ int quantity;

    
    if(threadIdx.x == 0){
        workSize = filA / gridDim.x;
        offset = filA % gridDim.x;
        quantity = (workSize + (blockIdx.x<offset));
        start = (blockIdx.x*workSize) + (blockIdx.x<offset?blockIdx.x:offset);
    }

    __syncthreads();

    for(int i= start; i < start+quantity; i++){
        for(int j=threadIdx.x; j < colB; j+=blockDim.x){
            for(int k=0; k < colA; k++){
                matC[i*colB+j] += matA[i*colA+k]*matB[k*colB+j];
            }
        }
    }
} 

int main(int argc, char* argv[]){
    if(argc != 4){
        printf("Usage: ./Ejercicio5 <filasA> <columnasA> <columnasB>\n");
        exit(EXIT_FAILURE);
    }

    int filA = atoi(argv[1]);
    int colA = atoi(argv[2]);
    int colB = atoi(argv[3]);

    double **matA = reservarMatriz(filA, colA);
    double **matB = reservarMatriz(colA, colB);
    double **matC = NULL;

    sillyInit(matA, filA, colA);
    sillyInit(matB, colA, colB);

    matC = productoMatrices(matA, filA, colA, matB, colB);

    if(filA < 7 && colB < 7){
        printf("Matriz A:\n");
        printMatrix(matA, filA, colA);
        printf("Matriz B:\n");
        printMatrix(matB, colA, colB);
        printf("MatrizCPU:\n");
        printMatrix(matC, filA, colB);
    }

    double* dev_matA = nullptr;
    double* dev_matB = nullptr;
    double* dev_matC = nullptr;
    double **matGPU = reservarMatriz(filA, colB);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliSeconds = 0.0;
    float aveGPUMS = 0.0;    

    for(int i=-1; i < numRuns; i++){ 
        hipEventRecord(start);

        hipMalloc( (void**) &dev_matA, filA*colA*sizeof(double));
        hipMemcpy( dev_matA, matA[0], filA*colA*sizeof(double), hipMemcpyHostToDevice);
        hipMalloc( (void**) &dev_matB, colB*colA*sizeof(double));
        hipMemcpy( dev_matB, matB[0], colB*colA*sizeof(double), hipMemcpyHostToDevice);
        hipMalloc( (void**) &dev_matC, filA*colB*sizeof(double)); 

        hipMemset(dev_matC, 0.0, filA*colB*sizeof(double));

        if(i < 0){
            kernelProductoMatrices<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_matA, filA, colA, dev_matB, colB, dev_matC);
        }

        else{
            hipDeviceSynchronize();
            kernelProductoMatrices<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_matA, filA, colA, dev_matB, colB, dev_matC);
            hipDeviceSynchronize();
            hipMemcpy( matGPU[0], dev_matC, filA*colB*sizeof(double), hipMemcpyDeviceToHost);
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliSeconds, start,stop);
        
        if(i >= 0)
            aveGPUMS+=milliSeconds;

        hipFree(dev_matA);
        hipFree(dev_matB);
        hipFree(dev_matC);

    }

    comprobarMatrices(matC, filA, colB, matGPU);
    printf("Tiempo usado por la GPU:%f\n",aveGPUMS/(1000*numRuns));

    if(filA < 7 && colB < 7){
        printf("\nMatrizGPU:\n");
        printMatrix(matGPU, filA, colB);
    }

    liberarMatriz(matA);
    liberarMatriz(matB);
    liberarMatriz(matC);
    liberarMatriz(matGPU);

    return 0;
}